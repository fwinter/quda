#include "hip/hip_runtime.h"
#include <read_gauge.h>
#include <gauge_field.h>

#include "gauge_force_quda.h"

#define MULT_SU3_NN_TEST(ma, mb) do{				\
    float fa_re,fa_im, fb_re, fb_im, fc_re, fc_im;		\
    fa_re =							\
      ma##00_re * mb##00_re - ma##00_im * mb##00_im +		\
	    ma##01_re * mb##10_re - ma##01_im * mb##10_im +	\
	    ma##02_re * mb##20_re - ma##02_im * mb##20_im;	\
	fa_im =							\
	    ma##00_re * mb##00_im + ma##00_im * mb##00_re +	\
	    ma##01_re * mb##10_im + ma##01_im * mb##10_re +	\
	    ma##02_re * mb##20_im + ma##02_im * mb##20_re;	\
	fb_re =							\
	    ma##00_re * mb##01_re - ma##00_im * mb##01_im +	\
	    ma##01_re * mb##11_re - ma##01_im * mb##11_im +	\
	    ma##02_re * mb##21_re - ma##02_im * mb##21_im;	\
	fb_im =							\
	    ma##00_re * mb##01_im + ma##00_im * mb##01_re +	\
	    ma##01_re * mb##11_im + ma##01_im * mb##11_re +	\
	    ma##02_re * mb##21_im + ma##02_im * mb##21_re;	\
	fc_re =							\
	    ma##00_re * mb##02_re - ma##00_im * mb##02_im +	\
	    ma##01_re * mb##12_re - ma##01_im * mb##12_im +	\
	    ma##02_re * mb##22_re - ma##02_im * mb##22_im;	\
	fc_im =							\
	    ma##00_re * mb##02_im + ma##00_im * mb##02_re +	\
	    ma##01_re * mb##12_im + ma##01_im * mb##12_re +	\
	    ma##02_re * mb##22_im + ma##02_im * mb##22_re;	\
	ma##00_re = fa_re;					\
	ma##00_im = fa_im;					\
	ma##01_re = fb_re;					\
	ma##01_im = fb_im;					\
	ma##02_re = fc_re;					\
	ma##02_im = fc_im;					\
	fa_re =							\
	    ma##10_re * mb##00_re - ma##10_im * mb##00_im +	\
	    ma##11_re * mb##10_re - ma##11_im * mb##10_im +	\
	    ma##12_re * mb##20_re - ma##12_im * mb##20_im;	\
	fa_im =							\
	    ma##10_re * mb##00_im + ma##10_im * mb##00_re +	\
	    ma##11_re * mb##10_im + ma##11_im * mb##10_re +	\
	    ma##12_re * mb##20_im + ma##12_im * mb##20_re;	\
	fb_re =							\
	    ma##10_re * mb##01_re - ma##10_im * mb##01_im +	\
	    ma##11_re * mb##11_re - ma##11_im * mb##11_im +	\
	    ma##12_re * mb##21_re - ma##12_im * mb##21_im;	\
	fb_im =							\
	    ma##10_re * mb##01_im + ma##10_im * mb##01_re +	\
	    ma##11_re * mb##11_im + ma##11_im * mb##11_re +	\
	    ma##12_re * mb##21_im + ma##12_im * mb##21_re;	\
	fc_re =							\
	    ma##10_re * mb##02_re - ma##10_im * mb##02_im +	\
	    ma##11_re * mb##12_re - ma##11_im * mb##12_im +	\
	    ma##12_re * mb##22_re - ma##12_im * mb##22_im;	\
	fc_im =							\
	    ma##10_re * mb##02_im + ma##10_im * mb##02_re +	\
	    ma##11_re * mb##12_im + ma##11_im * mb##12_re +	\
	    ma##12_re * mb##22_im + ma##12_im * mb##22_re;	\
	ma##10_re = fa_re;					\
	ma##10_im = fa_im;					\
	ma##11_re = fb_re;					\
	ma##11_im = fb_im;					\
	ma##12_re = fc_re;					\
	ma##12_im = fc_im;					\
	fa_re =							\
	    ma##20_re * mb##00_re - ma##20_im * mb##00_im +	\
	    ma##21_re * mb##10_re - ma##21_im * mb##10_im +	\
	    ma##22_re * mb##20_re - ma##22_im * mb##20_im;	\
	fa_im =							\
	    ma##20_re * mb##00_im + ma##20_im * mb##00_re +	\
	    ma##21_re * mb##10_im + ma##21_im * mb##10_re +	\
	    ma##22_re * mb##20_im + ma##22_im * mb##20_re;	\
	fb_re =							\
	    ma##20_re * mb##01_re - ma##20_im * mb##01_im +	\
	    ma##21_re * mb##11_re - ma##21_im * mb##11_im +	\
	    ma##22_re * mb##21_re - ma##22_im * mb##21_im;	\
	fb_im =							\
	    ma##20_re * mb##01_im + ma##20_im * mb##01_re +	\
	    ma##21_re * mb##11_im + ma##21_im * mb##11_re +	\
	    ma##22_re * mb##21_im + ma##22_im * mb##21_re;	\
	fc_re =							\
	    ma##20_re * mb##02_re - ma##20_im * mb##02_im +	\
	    ma##21_re * mb##12_re - ma##21_im * mb##12_im +	\
	    ma##22_re * mb##22_re - ma##22_im * mb##22_im;	\
	fc_im =							\
	    ma##20_re * mb##02_im + ma##20_im * mb##02_re +	\
	    ma##21_re * mb##12_im + ma##21_im * mb##12_re +	\
	    ma##22_re * mb##22_im + ma##22_im * mb##22_re;	\
	ma##20_re = fa_re;					\
	ma##20_im = fa_im;					\
	ma##21_re = fb_re;					\
	ma##21_im = fb_im;					\
	ma##22_re = fc_re;					\
	ma##22_im = fc_im;					\
    }while(0)


#define MULT_SU3_NA_TEST(ma, mb)	do{				\
	float fa_re, fa_im, fb_re, fb_im, fc_re, fc_im;			\
	fa_re =								\
	    ma##00_re * mb##T00_re - ma##00_im * mb##T00_im +		\
	    ma##01_re * mb##T10_re - ma##01_im * mb##T10_im +		\
	    ma##02_re * mb##T20_re - ma##02_im * mb##T20_im;		\
	fa_im =								\
	    ma##00_re * mb##T00_im + ma##00_im * mb##T00_re +		\
	    ma##01_re * mb##T10_im + ma##01_im * mb##T10_re +		\
	    ma##02_re * mb##T20_im + ma##02_im * mb##T20_re;		\
	fb_re =								\
	    ma##00_re * mb##T01_re - ma##00_im * mb##T01_im +		\
	    ma##01_re * mb##T11_re - ma##01_im * mb##T11_im +		\
	    ma##02_re * mb##T21_re - ma##02_im * mb##T21_im;		\
	fb_im =								\
	    ma##00_re * mb##T01_im + ma##00_im * mb##T01_re +		\
	    ma##01_re * mb##T11_im + ma##01_im * mb##T11_re +		\
	    ma##02_re * mb##T21_im + ma##02_im * mb##T21_re;		\
	fc_re =								\
	    ma##00_re * mb##T02_re - ma##00_im * mb##T02_im +		\
	    ma##01_re * mb##T12_re - ma##01_im * mb##T12_im +		\
	    ma##02_re * mb##T22_re - ma##02_im * mb##T22_im;		\
	fc_im =								\
	    ma##00_re * mb##T02_im + ma##00_im * mb##T02_re +		\
	    ma##01_re * mb##T12_im + ma##01_im * mb##T12_re +		\
	    ma##02_re * mb##T22_im + ma##02_im * mb##T22_re;		\
	ma##00_re = fa_re;						\
	ma##00_im = fa_im;						\
	ma##01_re = fb_re;						\
	ma##01_im = fb_im;						\
	ma##02_re = fc_re;						\
	ma##02_im = fc_im;						\
	fa_re =								\
	    ma##10_re * mb##T00_re - ma##10_im * mb##T00_im +		\
	    ma##11_re * mb##T10_re - ma##11_im * mb##T10_im +		\
	    ma##12_re * mb##T20_re - ma##12_im * mb##T20_im;		\
	fa_im =								\
	    ma##10_re * mb##T00_im + ma##10_im * mb##T00_re +		\
	    ma##11_re * mb##T10_im + ma##11_im * mb##T10_re +		\
	    ma##12_re * mb##T20_im + ma##12_im * mb##T20_re;		\
	fb_re =								\
	    ma##10_re * mb##T01_re - ma##10_im * mb##T01_im +		\
	    ma##11_re * mb##T11_re - ma##11_im * mb##T11_im +		\
	    ma##12_re * mb##T21_re - ma##12_im * mb##T21_im;		\
	fb_im =								\
	    ma##10_re * mb##T01_im + ma##10_im * mb##T01_re +		\
	    ma##11_re * mb##T11_im + ma##11_im * mb##T11_re +		\
	    ma##12_re * mb##T21_im + ma##12_im * mb##T21_re;		\
	fc_re =								\
	    ma##10_re * mb##T02_re - ma##10_im * mb##T02_im +		\
	    ma##11_re * mb##T12_re - ma##11_im * mb##T12_im +		\
	    ma##12_re * mb##T22_re - ma##12_im * mb##T22_im;		\
	fc_im =								\
	    ma##10_re * mb##T02_im + ma##10_im * mb##T02_re +		\
	    ma##11_re * mb##T12_im + ma##11_im * mb##T12_re +		\
	    ma##12_re * mb##T22_im + ma##12_im * mb##T22_re;		\
	ma##10_re = fa_re;						\
	ma##10_im = fa_im;						\
	ma##11_re = fb_re;						\
	ma##11_im = fb_im;						\
	ma##12_re = fc_re;						\
	ma##12_im = fc_im;						\
	fa_re =								\
	    ma##20_re * mb##T00_re - ma##20_im * mb##T00_im +		\
	    ma##21_re * mb##T10_re - ma##21_im * mb##T10_im +		\
	    ma##22_re * mb##T20_re - ma##22_im * mb##T20_im;		\
	fa_im =								\
	    ma##20_re * mb##T00_im + ma##20_im * mb##T00_re +		\
	    ma##21_re * mb##T10_im + ma##21_im * mb##T10_re +		\
	    ma##22_re * mb##T20_im + ma##22_im * mb##T20_re;		\
	fb_re =								\
	    ma##20_re * mb##T01_re - ma##20_im * mb##T01_im +		\
	    ma##21_re * mb##T11_re - ma##21_im * mb##T11_im +		\
	    ma##22_re * mb##T21_re - ma##22_im * mb##T21_im;		\
	fb_im =								\
	    ma##20_re * mb##T01_im + ma##20_im * mb##T01_re +		\
	    ma##21_re * mb##T11_im + ma##21_im * mb##T11_re +		\
	    ma##22_re * mb##T21_im + ma##22_im * mb##T21_re;		\
	fc_re =								\
	    ma##20_re * mb##T02_re - ma##20_im * mb##T02_im +		\
	    ma##21_re * mb##T12_re - ma##21_im * mb##T12_im +		\
	    ma##22_re * mb##T22_re - ma##22_im * mb##T22_im;		\
	fc_im =								\
	    ma##20_re * mb##T02_im + ma##20_im * mb##T02_re +		\
	    ma##21_re * mb##T12_im + ma##21_im * mb##T12_re +		\
	    ma##22_re * mb##T22_im + ma##22_im * mb##T22_re;		\
	ma##20_re = fa_re;						\
	ma##20_im = fa_im;						\
	ma##21_re = fb_re;						\
	ma##21_im = fb_im;						\
	ma##22_re = fc_re;						\
	ma##22_im = fc_im;						\
    }while(0)



#define MULT_SU3_AN_TEST(ma, mb)	do{				\
	float fa_re, fa_im, fb_re, fb_im, fc_re, fc_im;			\
	fa_re =								\
	    ma##T00_re * mb##00_re - ma##T00_im * mb##00_im +		\
	    ma##T01_re * mb##10_re - ma##T01_im * mb##10_im +		\
	    ma##T02_re * mb##20_re - ma##T02_im * mb##20_im;		\
	fa_im =								\
	    ma##T00_re * mb##00_im + ma##T00_im * mb##00_re +		\
	    ma##T01_re * mb##10_im + ma##T01_im * mb##10_re +		\
	    ma##T02_re * mb##20_im + ma##T02_im * mb##20_re;		\
	fb_re =								\
	    ma##T10_re * mb##00_re - ma##T10_im * mb##00_im +		\
	    ma##T11_re * mb##10_re - ma##T11_im * mb##10_im +		\
	    ma##T12_re * mb##20_re - ma##T12_im * mb##20_im;		\
	fb_im =								\
	    ma##T10_re * mb##00_im + ma##T10_im * mb##00_re +		\
	    ma##T11_re * mb##10_im + ma##T11_im * mb##10_re +		\
	    ma##T12_re * mb##20_im + ma##T12_im * mb##20_re;		\
	fc_re =								\
	    ma##T20_re * mb##00_re - ma##T20_im * mb##00_im +		\
	    ma##T21_re * mb##10_re - ma##T21_im * mb##10_im +		\
	    ma##T22_re * mb##20_re - ma##T22_im * mb##20_im;		\
	fc_im =								\
	    ma##T20_re * mb##00_im + ma##T20_im * mb##00_re +		\
	    ma##T21_re * mb##10_im + ma##T21_im * mb##10_re +		\
	    ma##T22_re * mb##20_im + ma##T22_im * mb##20_re;		\
	mb##00_re = fa_re;						\
	mb##00_im = fa_im;						\
	mb##10_re = fb_re;						\
	mb##10_im = fb_im;						\
	mb##20_re = fc_re;						\
	mb##20_im = fc_im;						\
	fa_re =								\
	    ma##T00_re * mb##01_re - ma##T00_im * mb##01_im +		\
	    ma##T01_re * mb##11_re - ma##T01_im * mb##11_im +		\
	    ma##T02_re * mb##21_re - ma##T02_im * mb##21_im;		\
	fa_im =								\
	    ma##T00_re * mb##01_im + ma##T00_im * mb##01_re +		\
	    ma##T01_re * mb##11_im + ma##T01_im * mb##11_re +		\
	    ma##T02_re * mb##21_im + ma##T02_im * mb##21_re;		\
	fb_re =								\
	    ma##T10_re * mb##01_re - ma##T10_im * mb##01_im +		\
	    ma##T11_re * mb##11_re - ma##T11_im * mb##11_im +		\
	    ma##T12_re * mb##21_re - ma##T12_im * mb##21_im;		\
	fb_im =								\
	    ma##T10_re * mb##01_im + ma##T10_im * mb##01_re +		\
	    ma##T11_re * mb##11_im + ma##T11_im * mb##11_re +		\
	    ma##T12_re * mb##21_im + ma##T12_im * mb##21_re;		\
	fc_re =								\
	    ma##T20_re * mb##01_re - ma##T20_im * mb##01_im +		\
	    ma##T21_re * mb##11_re - ma##T21_im * mb##11_im +		\
	    ma##T22_re * mb##21_re - ma##T22_im * mb##21_im;		\
	fc_im =								\
	    ma##T20_re * mb##01_im + ma##T20_im * mb##01_re +		\
	    ma##T21_re * mb##11_im + ma##T21_im * mb##11_re +		\
	    ma##T22_re * mb##21_im + ma##T22_im * mb##21_re;		\
	mb##01_re = fa_re;						\
	mb##01_im = fa_im;						\
	mb##11_re = fb_re;						\
	mb##11_im = fb_im;						\
	mb##21_re = fc_re;						\
	mb##21_im = fc_im;						\
	fa_re =								\
	    ma##T00_re * mb##02_re - ma##T00_im * mb##02_im +		\
	    ma##T01_re * mb##12_re - ma##T01_im * mb##12_im +		\
	    ma##T02_re * mb##22_re - ma##T02_im * mb##22_im;		\
	fa_im =								\
	    ma##T00_re * mb##02_im + ma##T00_im * mb##02_re +		\
	    ma##T01_re * mb##12_im + ma##T01_im * mb##12_re +		\
	    ma##T02_re * mb##22_im + ma##T02_im * mb##22_re;		\
	fb_re =								\
	    ma##T10_re * mb##02_re - ma##T10_im * mb##02_im +		\
	    ma##T11_re * mb##12_re - ma##T11_im * mb##12_im +		\
	    ma##T12_re * mb##22_re - ma##T12_im * mb##22_im;		\
	fb_im =								\
	    ma##T10_re * mb##02_im + ma##T10_im * mb##02_re +		\
	    ma##T11_re * mb##12_im + ma##T11_im * mb##12_re +		\
	    ma##T12_re * mb##22_im + ma##T12_im * mb##22_re;		\
	fc_re =								\
	    ma##T20_re * mb##02_re - ma##T20_im * mb##02_im +		\
	    ma##T21_re * mb##12_re - ma##T21_im * mb##12_im +		\
	    ma##T22_re * mb##22_re - ma##T22_im * mb##22_im;		\
	fc_im =								\
	    ma##T20_re * mb##02_im + ma##T20_im * mb##02_re +		\
	    ma##T21_re * mb##12_im + ma##T21_im * mb##12_re +		\
	    ma##T22_re * mb##22_im + ma##T22_im * mb##22_re;		\
	mb##02_re = fa_re;						\
	mb##02_im = fa_im;						\
	mb##12_re = fb_re;						\
	mb##12_im = fb_im;						\
	mb##22_re = fc_re;						\
	mb##22_im = fc_im;						\
    }while(0)


#define GF_SITE_MATRIX_LOAD_TEX 1

#if (GF_SITE_MATRIX_LOAD_TEX == 1)

#define LOAD_EVEN_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE_TEX(siteLink0TexSingle_recon, dir, idx, var)
#define LOAD_ODD_MATRIX(dir, idx, var) 	LOAD_MATRIX_12_SINGLE_TEX(siteLink1TexSingle_recon, dir, idx, var)
#else
#define LOAD_EVEN_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE(linkEven, dir, idx, var)
#define LOAD_ODD_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE(linkOdd, dir, idx, var)
#endif


#define LOAD_MATRIX LOAD_MATRIX_12_SINGLE
#define LOAD_ANTI_HERMITIAN LOAD_ANTI_HERMITIAN_SINGLE
#define WRITE_ANTI_HERMITIAN WRITE_ANTI_HERMITIAN_SINGLE
#define RECONSTRUCT_MATRIX RECONSTRUCT_LINK_12


__constant__ int path_max_length;

void
gauge_force_init_cuda(QudaGaugeParam* param, int path_max_length)
{    
  
#ifdef MULTI_GPU
#error "multi gpu is not supported for gauge force computation"  
#endif
  
    static int gauge_force_init_cuda_flag = 0;
    if (gauge_force_init_cuda_flag){
	return;
    }
    gauge_force_init_cuda_flag=1;

    init_kernel_cuda(param);
    
    hipMemcpyToSymbol(HIP_SYMBOL("path_max_length"), &path_max_length, sizeof(int));

}

#define COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(mydir, idx) do {		\
        switch(mydir){                                                  \
        case 0:                                                         \
            new_mem_idx = ( (new_x1==X1m1)?idx-X1m1:idx+1);		\
	    new_x1 = (new_x1==X1m1)?0:new_x1+1;				\
            break;                                                      \
        case 1:                                                         \
            new_mem_idx = ( (new_x2==X2m1)?idx-X2X1mX1:idx+X1);		\
	    new_x2 = (new_x2==X2m1)?0:new_x2+1;				\
            break;                                                      \
        case 2:                                                         \
            new_mem_idx = ( (new_x3==X3m1)?idx-X3X2X1mX2X1:idx+X2X1);	\
	    new_x3 = (new_x3==X3m1)?0:new_x3+1;				\
            break;                                                      \
        case 3:                                                         \
            new_mem_idx = ( (new_x4==X4m1)?idx-X4X3X2X1mX3X2X1:idx+X3X2X1); \
	    new_x4 = (new_x4==X4m1)?0:new_x4+1;				\
            break;                                                      \
        }                                                               \
    }while(0)

#define COMPUTE_NEW_FULL_IDX_MINUS_UPDATE(mydir, idx) do {		\
        switch(mydir){                                                  \
        case 0:                                                         \
            new_mem_idx = ( (new_x1==0)?idx+X1m1:idx-1);		\
	    new_x1 = (new_x1==0)?X1m1:new_x1 - 1;			\
            break;                                                      \
        case 1:                                                         \
            new_mem_idx = ( (new_x2==0)?idx+X2X1mX1:idx-X1);		\
	    new_x2 = (new_x2==0)?X2m1:new_x2 - 1;			\
            break;                                                      \
        case 2:                                                         \
            new_mem_idx = ( (new_x3==0)?idx+X3X2X1mX2X1:idx-X2X1);	\
	    new_x3 = (new_x3==0)?X3m1:new_x3 - 1;			\
            break;                                                      \
        case 3:                                                         \
            new_mem_idx = ( (new_x4==0)?idx+X4X3X2X1mX3X2X1:idx-X3X2X1); \
	    new_x4 = (new_x4==0)?X4m1:new_x4 - 1;			\
            break;                                                      \
        }                                                               \
    }while(0)



//for now we only consider 12-reconstruct and single precision

template<int oddBit>
__global__ void
parity_compute_gauge_force_kernel(float2* momEven, float2* momOdd,
				  int dir, double eb3,
				  float4* linkEven, float4* linkOdd,
				  int* input_path, 
				  int* length, float* path_coeff, int num_paths)
{
    int i,j=0;
    int sid = blockIdx.x * blockDim.x + threadIdx.x;
    
    int z1 = sid / X1h;
    int x1h = sid - z1*X1h;
    int z2 = z1 / X2;
    int x2 = z1 - z2*X2;
    int x4 = z2 / X3;
    int x3 = z2 - x4*X3;
    int x1odd = (x2 + x3 + x4 + oddBit) & 1;
    int x1 = 2*x1h + x1odd;  
    int X = 2*sid + x1odd;
    
    const int sign = 1;
    
    float2* mymom=momEven;
    if (oddBit){
	mymom = momOdd;
    }

    float4 LINKA0, LINKA1, LINKA2, LINKA3, LINKA4;
    float4 LINKB0, LINKB1, LINKB2, LINKB3, LINKB4;
    float2 STAPLE0, STAPLE1, STAPLE2, STAPLE3,STAPLE4, STAPLE5, STAPLE6, STAPLE7, STAPLE8;
    float2 AH0, AH1, AH2, AH3, AH4;

    int new_mem_idx;
    
    
    SET_SU3_MATRIX(staple, 0);
    for(i=0;i < num_paths; i++){
	int nbr_oddbit = (oddBit^1 );
	
	int new_x1 =x1;
	int new_x2 =x2;
	int new_x3 =x3;
	int new_x4 =x4;
	COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(dir, X);
	
	//linka: current matrix
	//linkb: the loaded matrix in this round	
	SET_UNIT_SU3_MATRIX(linka);	
	int* path = input_path + i*path_max_length;
	
	int lnkdir;
	int path0 = path[0];
	if (GOES_FORWARDS(path0)){
	    lnkdir=path0;
	}else{
	    lnkdir=OPP_DIR(path0);
	    COMPUTE_NEW_FULL_IDX_MINUS_UPDATE(OPP_DIR(path0), new_mem_idx);
	    nbr_oddbit = nbr_oddbit^1;
	    
	}
	
	int nbr_idx = new_mem_idx >>1;
	if (nbr_oddbit){
	    LOAD_ODD_MATRIX( lnkdir, nbr_idx, LINKB);
	}else{
	    LOAD_EVEN_MATRIX( lnkdir, nbr_idx, LINKB);
	}
	
	RECONSTRUCT_MATRIX(lnkdir, nbr_idx, sign, linkb);
	if (GOES_FORWARDS(path0)){
	    COPY_SU3_MATRIX(linkb, linka);
	    COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(path0, new_mem_idx);
	    nbr_oddbit = nbr_oddbit^1;
	}else{
	    SU3_ADJOINT(linkb, linka);
	}	
	
	for(j=1; j < length[i]; j++){
	    
	    int lnkdir;
	    int pathj = path[j];
	    if (GOES_FORWARDS(pathj)){
		lnkdir=pathj;
	    }else{
		lnkdir=OPP_DIR(pathj);
		COMPUTE_NEW_FULL_IDX_MINUS_UPDATE(OPP_DIR(pathj), new_mem_idx);
		nbr_oddbit = nbr_oddbit^1;

	    }
	    
	    int nbr_idx = new_mem_idx >>1;
	    if (nbr_oddbit){
		LOAD_ODD_MATRIX(lnkdir, nbr_idx, LINKB);
	    }else{
		LOAD_EVEN_MATRIX(lnkdir, nbr_idx, LINKB);
	    }
	    RECONSTRUCT_MATRIX(lnkdir, nbr_idx, sign, linkb);
	    if (GOES_FORWARDS(pathj)){
	      MULT_SU3_NN_TEST(linka, linkb);
		
		COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(pathj, new_mem_idx);
		nbr_oddbit = nbr_oddbit^1;
		
		
	    }else{
		MULT_SU3_NA_TEST(linka, linkb);		
	    }
	    
	}//j
	SCALAR_MULT_ADD_SU3_MATRIX(staple, linka, path_coeff[i], staple);
    }//i
    

    //update mom 
    if (oddBit){
	LOAD_ODD_MATRIX(dir, sid, LINKA);
    }else{
	LOAD_EVEN_MATRIX(dir, sid, LINKA);
    }
    RECONSTRUCT_MATRIX(dir, sid, sign, linka);
    MULT_SU3_NN_TEST(linka, staple);
    LOAD_ANTI_HERMITIAN(mymom, dir, sid, AH);
    UNCOMPRESS_ANTI_HERMITIAN(ah, linkb);
    SCALAR_MULT_SUB_SU3_MATRIX(linkb, linka, eb3, linka);
    MAKE_ANTI_HERMITIAN(linka, ah);
    
    WRITE_ANTI_HERMITIAN(mymom, dir, sid, AH);

    return;
}

void
gauge_force_cuda(cudaGaugeField&  cudaMom, int dir, double eb3, cudaGaugeField& cudaSiteLink,
                 QudaGaugeParam* param, int** input_path, 
		 int* length, void* path_coeff, int num_paths, int max_length)
{

    int i, j;
    //input_path
    int bytes = num_paths*max_length* sizeof(int);
    int* input_path_d;
    hipMalloc((void**)&input_path_d, bytes); checkCudaError();    
    hipMemset(input_path_d, 0, bytes);checkCudaError();

    int* input_path_h = (int*)malloc(bytes);
    if (input_path_h == NULL){
	printf("ERROR: malloc failed for input_path_h in function %s\n", __FUNCTION__);
	exit(1);
    }
        
    memset(input_path_h, 0, bytes);
    for(i=0;i < num_paths;i++){
	for(j=0; j < length[i]; j++){
	    input_path_h[i*max_length + j] =input_path[i][j];
	}
    }

    hipMemcpy(input_path_d, input_path_h, bytes, hipMemcpyHostToDevice); checkCudaError();
    
    //length
    int* length_d;
    hipMalloc((void**)&length_d, num_paths*sizeof(int)); checkCudaError();
    hipMemcpy(length_d, length, num_paths*sizeof(int), hipMemcpyHostToDevice); checkCudaError();
    
    //path_coeff
    int gsize;
    if (param->cuda_prec == QUDA_DOUBLE_PRECISION){
	gsize = sizeof(double);
    }else{
	gsize= sizeof(float);
    }     
    void* path_coeff_d;
    hipMalloc((void**)&path_coeff_d, num_paths*gsize); checkCudaError();
    hipMemcpy(path_coeff_d, path_coeff, num_paths*gsize, hipMemcpyHostToDevice); checkCudaError();

    //compute the gauge forces
    int volume = param->X[0]*param->X[1]*param->X[2]*param->X[3];
    dim3 blockDim(BLOCK_DIM, 1,1);
    dim3 gridDim(volume/blockDim.x, 1, 1);
    dim3 halfGridDim(volume/(2*blockDim.x), 1, 1);
    
    float2* momEven = (float2*)cudaMom.Even_p();
    float2* momOdd = (float2*)cudaMom.Odd_p();
    float4* linkEven = (float4*)cudaSiteLink.Even_p();
    float4* linkOdd = (float4*)cudaSiteLink.Odd_p();        

    hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.Even_p(), cudaSiteLink.Bytes());
    hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.Odd_p(), cudaSiteLink.Bytes());
    parity_compute_gauge_force_kernel<0><<<halfGridDim, blockDim>>>(momEven, momOdd,
								    dir, eb3,
								    linkEven, linkOdd, 
								    input_path_d, length_d, (float*)path_coeff_d,
								    num_paths);   
    //odd
    /* The reason we do not switch the even/odd function input paramemters and the texture binding
     * is that we use the oddbit to decided where to load, in the kernel function
     */
    parity_compute_gauge_force_kernel<1><<<halfGridDim, blockDim>>>(momEven, momOdd,
								  dir, eb3,
								  linkEven, linkOdd, 
								  input_path_d, length_d, (float*)path_coeff_d,
								  num_paths);  
    

    
    hipUnbindTexture(siteLink0TexSingle_recon);
    hipUnbindTexture(siteLink1TexSingle_recon);
    
    checkCudaError();
    
    hipFree(input_path_d); checkCudaError();
    free(input_path_h);
    hipFree(length_d);
    hipFree(path_coeff_d);

    

}


#undef LOAD_EVEN_MATRIX
#undef LOAD_ODD_MATRIX
#undef LOAD_MATRIX 
#undef LOAD_ANTI_HERMITIAN 
#undef WRITE_ANTI_HERMITIAN
#undef RECONSTRUCT_MATRIX
